#include <cstddef>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>
#include <cstdlib>

using namespace std;

#define M 1024
#define N 1024
#define K 1024
#define BLOCKDIM 32
#define PATCH (TILEDIM / COARSENING_FACTOR)  // 同一个thread处理的元素之间间隔的行数

// A: M * K; B: K * N
template<size_t TILEDIM, size_t COARSENING_FACTOR>
__global__ void gemm2_register(float *a, float *b, float *c) {
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    int grow = by * blockDim.y * COARSENING_FACTOR + ty;
    int gcol = bx * blockDim.x + tx;
    __shared__ float tileA[TILEDIM][TILEDIM];
    __shared__ float tileB[TILEDIM][TILEDIM];

    int phase = ceil(1.f * K / TILEDIM);
    float pval[COARSENING_FACTOR] = {0.f};
    for (int i = 0; i < phase; i++) {
        // global -> shared: load tile
        #pragma unroll
        for (int j = 0; j < COARSENING_FACTOR; j++) {
            if (grow + j * PATCH < M && i * TILEDIM + tx < K)
                tileA[ty + j * PATCH][tx] = a[(grow + j * PATCH) * K + i * TILEDIM + tx];
            else
                tileA[ty + j * PATCH][tx] = 0.f;
        }
        #pragma unroll
        for (int j = 0; j < COARSENING_FACTOR; j++) {
            if (i * TILEDIM + ty + j * PATCH < K && gcol < N)
                tileB[ty + j * PATCH][tx] = b[(i * TILEDIM + ty + j * PATCH) * N + gcol];
            else 
                tileB[ty + j * PATCH][tx] = 0.f;
        }
        __syncthreads();

        // partial dot product
        for (int k = 0; k < TILEDIM; k++) {
            float reg_b = tileB[k][tx];
            #pragma unroll
            for (int p = 0; p < COARSENING_FACTOR; p++) {
                pval[p] += tileA[ty + p * PATCH][k] * reg_b;  // register value can be reused multiple times
            }
        }
        __syncthreads();
    }

    #pragma unroll
    for (int i = 0; i < COARSENING_FACTOR; i++) {
        if (grow + i * PATCH < M && gcol < N)
            c[(grow + i * PATCH) * N + gcol] = pval[i];
    }
}

float* init(float *a, float *b) {
    for (int i = 0; i < M * N; i++) {
        a[i] = 1.f;
        b[i] = 1.f;
    }
    float *c = (float *) calloc(M * N, sizeof(float));
    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++)
            for (int k = 0; k < K; k++)
                c[i * N + j] += a[i * K + k] * b[k * N + j];
    
    return c;
}

bool check_ans(float *truth, float *c) {
    for (int i = 0; i < M * N; i++)
        if (fabs(c[i] - truth[i]) > 0.5) {
            printf("i: %d, truth: %f, output: %f\n", i, truth[i], c[i]);
            return false;
        }
    return true;
}

int main() {
    size_t size = sizeof(float) * M * N;
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    float *c_truth = init(a_h, b_h);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed = 0.f;
    const int nWarmup = 2;
    const int nIters = 3;

    const size_t TILEDIM = 32;
    const size_t COARSENING_FACTOR = 8;
    dim3 gridDim(ceil(1.f * M / BLOCKDIM), ceil(1.f * N / BLOCKDIM), 1);
    dim3 blockDim(BLOCKDIM, BLOCKDIM / COARSENING_FACTOR, 1);  // block 的y维缩小COARSENING_FACTOR倍

    for (int i = 0; i < nIters + nWarmup; i++) {
        hipEventRecord(start);
        gemm2_register<TILEDIM, COARSENING_FACTOR><<<gridDim, blockDim>>>(a_d, b_d, c_d);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i < nWarmup) {
            if (i == 0) {
                hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
                if (!check_ans(c_truth, c_h)) {
                    cerr << "result is wrong!" << endl;
                    return -1;
                }
                cout << "result is right" << endl;
            }
        } else {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsed += ms;
        }
    }
    const int64_t flop = int64_t(M) * int64_t(N) * int64_t(K) * 2;
    double gflops = flop / ((elapsed / nIters) / 1000) / 1e9;
    cout << "kernel: " << gflops << "GFLOPS (" << flop << " flop, " << (elapsed / nIters) / 1000 << "s)\n";

    return 0;
}
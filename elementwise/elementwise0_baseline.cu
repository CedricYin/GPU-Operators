#include <cstddef>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <cfloat>
#include <hip/hip_runtime.h>


using namespace std;

#define N 1024 * 1024 * 32
#define BLOCKDIM 1024

__global__ void elementwise0_baseline(float *a, float *b, float *c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

bool check_ans(float *arr) {
    for (int i = 0; i < N; i++)
        if (fabs(arr[i] - 3.f) > FLT_EPSILON)
            return false;
    return true;
}

int main() {
    size_t size = N * sizeof(float);
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    for (int i = 0; i < N; i++) {
        a_h[i] = 1.f;
        b_h[i] = 2.f;
    }

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    dim3 gridDim(ceil(1.f * N / BLOCKDIM));
    dim3 blockDim(BLOCKDIM);
    hipEventRecord(start);
    elementwise0_baseline<<<gridDim, blockDim>>>(a_d, b_d, c_d);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    if (!check_ans(c_h)) {
        cerr << "answer is wrong!" << endl;
        return -1;
    }

    cout << "elapsedTime: " << elapsedTime * 1000 << " ns" << endl;
    return 0;
}
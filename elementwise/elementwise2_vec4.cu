#include <cstddef>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <cfloat>
#include <hip/hip_runtime.h>


using namespace std;

#define N 1024 * 1024 * 32
#define BLOCKDIM 1024
#define VECTOR_FACTOR 4

#define FETCH_VEC4(ptr) (((float4 *) (ptr))[0])

__global__ void elementwise2_vec4(float *a, float *b, float *c) {
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * VECTOR_FACTOR;
    float4 vec2_a = FETCH_VEC4(&a[i]);
    float4 vec2_b = FETCH_VEC4(&b[i]);
    float4 vec2_c;
    vec2_c.x = vec2_a.x + vec2_b.x;
    vec2_c.y = vec2_a.y + vec2_b.y;
    vec2_c.z = vec2_a.z + vec2_b.z;
    vec2_c.w = vec2_a.w + vec2_b.w;
    FETCH_VEC4(&c[i]) = vec2_c;
}

bool check_ans(float *arr) {
    for (int i = 0; i < N; i++)
        if (fabs(arr[i] - 3.f) > FLT_EPSILON)
            return false;
    return true;
}

int main() {
    size_t size = N * sizeof(float);
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    for (int i = 0; i < N; i++) {
        a_h[i] = 1.f;
        b_h[i] = 2.f;
    }

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    dim3 gridDim(ceil(1.f * N / BLOCKDIM / VECTOR_FACTOR));
    dim3 blockDim(BLOCKDIM);
    hipEventRecord(start);
    elementwise2_vec4<<<gridDim, blockDim>>>(a_d, b_d, c_d);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    if (!check_ans(c_h)) {
        cerr << "answer is wrong!" << endl;
        return -1;
    }

    cout << "elapsedTime: " << elapsedTime * 1000 << " ns" << endl;
    return 0;
}
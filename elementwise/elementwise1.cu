#include <cstddef>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <cfloat>
#include <hip/hip_runtime.h>


using namespace std;

#define N 8192*8192
#define BLOCKDIM 1024
#define VECTOR_FACTOR 2

#define FETCH_VEC2(ptr) (((float2 *) (ptr))[0])

__global__ void elementwise1(const float *__restrict__ a, const float *__restrict__ b, float *c) {
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * VECTOR_FACTOR;
    float2 vec2_a = FETCH_VEC2(&a[i]);
    float2 vec2_b = FETCH_VEC2(&b[i]);
    float2 vec2_c;
    vec2_c.x = vec2_a.x + vec2_b.x;
    vec2_c.y = vec2_a.y + vec2_b.y;
    FETCH_VEC2(&c[i]) = vec2_c;
}

bool check_ans(float *arr) {
    for (int i = 0; i < N; i++)
        if (fabs(arr[i] - 3.f) > FLT_EPSILON)
            return false;
    return true;
}

int main() {
    size_t size = N * sizeof(float);
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    for (int i = 0; i < N; i++) {
        a_h[i] = 1.f;
        b_h[i] = 2.f;
    }

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    dim3 gridDim(ceil(1.f * N / BLOCKDIM / VECTOR_FACTOR));
    dim3 blockDim(BLOCKDIM);
    const int nWarmup = 2;
    const int nIter = 10;
    float elapsedTime = 0;
    for (int i = 0; i < nWarmup + nIter; i++) {
        hipEventRecord(start);
        elementwise1<<<gridDim, blockDim>>>(a_d, b_d, c_d);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i == 0) {
            hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
            if (!check_ans(c_h)) {
                cerr << "answer is wrong!" << endl;
                return -1;
            }
        }
        if (i >= nWarmup) {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsedTime += ms;
        }
    }

    const unsigned int nbytes = 3 * N * 4;
    double bw = 1. * bw / (elapsedTime / nIter / 1000) / 1e9;
    cout << "effective bandwidth: " << bw << "GB/s\n";
    cout << "% of V100 peak bandwidth: " << bw / 900 * 100 << "%" << endl;
    return 0;
}
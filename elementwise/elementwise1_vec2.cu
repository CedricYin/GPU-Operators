#include <cstddef>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <cfloat>
#include <hip/hip_runtime.h>


using namespace std;

#define N 1024 * 1024 * 32
#define BLOCKDIM 1024
#define VECTOR_FACTOR 2

#define FETCH_VEC2(ptr) (((float2 *) (ptr))[0])

__global__ void elementwise1_vec2(float *a, float *b, float *c) {
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * VECTOR_FACTOR;
    float2 vec2_a = FETCH_VEC2(&a[i]);
    float2 vec2_b = FETCH_VEC2(&b[i]);
    float2 vec2_c;
    vec2_c.x = vec2_a.x + vec2_b.x;
    vec2_c.y = vec2_a.y + vec2_b.y;
    FETCH_VEC2(&c[i]) = vec2_c;
}

bool check_ans(float *arr) {
    for (int i = 0; i < N; i++)
        if (fabs(arr[i] - 3.f) > FLT_EPSILON)
            return false;
    return true;
}

int main() {
    size_t size = N * sizeof(float);
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    for (int i = 0; i < N; i++) {
        a_h[i] = 1.f;
        b_h[i] = 2.f;
    }

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    dim3 gridDim(ceil(1.f * N / BLOCKDIM / VECTOR_FACTOR));
    dim3 blockDim(BLOCKDIM);
    hipEventRecord(start);
    elementwise1_vec2<<<gridDim, blockDim>>>(a_d, b_d, c_d);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    if (!check_ans(c_h)) {
        cerr << "answer is wrong!" << endl;
        return -1;
    }

    cout << "elapsedTime: " << elapsedTime * 1000 << " ns" << endl;
    return 0;
}
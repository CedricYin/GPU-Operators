#include <cstddef>
#include <iostream>
#include <hip/hip_runtime.h>

#include <cfloat>
#include <cstdlib>

using namespace std;

#define checkCudaErrors(func)				                                            \
{									                                                    \
    hipError_t e = (func);			                                                    \
    if(e != hipSuccess)						                                        \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}


template<int TILEDIM>
__global__ void transpose3_shared(float *input, float *output, size_t rows, size_t cols) {
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int igy_start = blockDim.y * blockIdx.y;
    int igx_start = blockDim.x * blockIdx.x;
    int ogy_start = blockDim.y * blockIdx.x;
    int ogx_start = blockDim.x * blockIdx.y;
    __shared__ float tile[TILEDIM][TILEDIM];

    if (igy_start + ty < rows && igx_start + tx < cols) {
        tile[ty][tx] = input[(igy_start + ty) * cols + (igx_start + tx)];
        __syncthreads();

        output[(ogy_start + ty) * rows + (ogx_start + tx)] = tile[tx][ty];  // 转置后的leading dimension变成了rows
    }
}

static float *truth;
bool check_ans(float *arr, size_t size) {
    for (int i = 0; i < size; i++) {
        if (fabs(truth[i] - arr[i]) > FLT_EPSILON) {
            return false;
        }
    }
    return true;
}

int main() {
    const int ROWS = 4096;
    const int COLS = 4096 * 32;
    const int N = ROWS * COLS;
    const int BLOCKDIM = 32;
    const int TILEDIM = 32;
    const size_t accessed_bytes = N * sizeof(float) * 2;

    size_t size = N * sizeof(float);
    float *a_h = (float *) malloc(size);
    for (int i = 0; i < N; i++) {
        a_h[i] = 1.f * i;
    }
    float *b_h = (float *) malloc(size);
    truth = (float *) malloc(size);
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++)
            truth[j * ROWS + i] = a_h[i * COLS + j];
    }

    float *a_d, *b_d;
    checkCudaErrors(hipMalloc((void **) &a_d, size));
    checkCudaErrors(hipMalloc((void **) &b_d, size));
    checkCudaErrors(hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice));
    
    dim3 gridDim(ceil(1.f * COLS / BLOCKDIM), ceil(1.f * ROWS / BLOCKDIM), 1);
    dim3 blockDim(BLOCKDIM, BLOCKDIM, 1);
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    const int nWarmup = 1;
    const int nIter = 10;
    float elapsed = 0.f;

    for (int i = 0; i < nWarmup + nIter; i++) {
        checkCudaErrors(hipEventRecord(start));
        transpose3_shared<TILEDIM><<<gridDim, blockDim>>>(a_d, b_d, ROWS, COLS);
        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        if (i < nWarmup) {
            if (i == 0) {
                checkCudaErrors(hipMemcpy(b_h, b_d, size, hipMemcpyDeviceToHost));
                if (!check_ans(b_h, N)) {
                    cerr << "result is wrong!" << endl;
                    return -1;
                }
                cout << "result is right" << endl;
            }
        } else {
            float ms;
            checkCudaErrors(hipEventElapsedTime(&ms, start, stop));
            elapsed += ms;
        }
    }
    cout << "average elapsed time: " << elapsed / nIter << "ms\n";
    double bw = accessed_bytes / (elapsed / nIter / 1000) / 1e9;
    cout << "bandwidth: " << bw << "GB/s, " << bw / 900 * 100 << "% of peak bandwidth" << endl;

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipFree(a_d));
    checkCudaErrors(hipFree(b_d));
    free(a_h);
    free(b_h);
    free(truth);
    return 0;
}
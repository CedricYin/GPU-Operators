#include <cstddef>
#include <iostream>
#include <hip/hip_runtime.h>

#include <cfloat>
#include <cstdlib>
#include <hipblas.h>

using namespace std;

#define checkCudaErrors(func)				                                            \
{									                                                    \
    hipError_t e = (func);			                                                    \
    if(e != hipSuccess)						                                        \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

static float *truth;
bool check_ans(float *arr, size_t size) {
    for (int i = 0; i < size; i++) {
        if (fabs(truth[i] - arr[i]) > FLT_EPSILON) {
            return false;
        }
    }
    return true;
}

int main() {
    const int ROWS = 4096;
    const int COLS = 4096 * 32;
    const int N = ROWS * COLS;
    const size_t accessed_bytes = N * sizeof(float) * 2;

    size_t size = N * sizeof(float);
    float *a_h = (float *) malloc(size);
    for (int i = 0; i < N; i++) {
        a_h[i] = 1.f * i;
    }
    float *a_clone = (float *) malloc(N * sizeof(float));
    memcpy(a_clone, a_h, N * sizeof(float));
    
    float *b_h = (float *) malloc(size);
    truth = (float *) malloc(size);
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++)
            truth[j * ROWS + i] = a_h[i * COLS + j];
    }

    float *a_d, *b_d;
    checkCudaErrors(hipMalloc((void **) &a_d, size));
    checkCudaErrors(hipMalloc((void **) &b_d, size));
    checkCudaErrors(hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    const int nWarmup = 1;
    const int nIter = 10;
    float elapsed = 0.f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.f;
    float beta = 0.f;

    for (int i = 0; i < nWarmup + nIter; i++) {
        checkCudaErrors(hipEventRecord(start));
        hipblasStatus_t e = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ROWS, COLS, &alpha, a_d, COLS, &beta, NULL, COLS, b_d, ROWS);
        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        if (e == HIPBLAS_STATUS_INVALID_VALUE) {
            std::cout << "CUBLAS_STATUS_INVALID_VALUE" << std::endl;
            return -1;
        }
        if (i < nWarmup) {
            if (i == 0) {
                checkCudaErrors(hipMemcpy(b_h, b_d, size, hipMemcpyDeviceToHost));
                if (!check_ans(b_h, N)) {
                    cerr << "result is wrong!" << endl;
                    return -1;
                }
                cout << "result is right" << endl;
            }
        } else {
            float ms;
            checkCudaErrors(hipEventElapsedTime(&ms, start, stop));
            elapsed += ms;
        }
    }
    cout << "average elapsed time: " << elapsed / nIter << "ms\n";
    double bw = accessed_bytes / (elapsed / nIter / 1000) / 1e9;
    cout << "bandwidth: " << bw << "GB/s, " << bw / 900 * 100 << "% of peak bandwidth" << endl;

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipFree(a_d));
    checkCudaErrors(hipFree(b_d));
    free(a_h);
    free(b_h);
    free(truth);
    return 0;
}
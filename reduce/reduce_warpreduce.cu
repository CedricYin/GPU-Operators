#include <cassert>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>


#define N 256 * 1024 * 1024
#define BLOCKDIM 1024
#define COARSE_FACTOR 2

using namespace std;

__device__ void warp_reduce(volatile int *input, unsigned tid) {
    input[tid] += input[tid + 32];
    input[tid] += input[tid + 16];
    input[tid] += input[tid + 8];
    input[tid] += input[tid + 4];
    input[tid] += input[tid + 2];
    input[tid] += input[tid + 1];
}

// using warp reduce
__global__ void reduce_warp_reduce(int *input, int *output) {
    extern __shared__ int input_s[];
    unsigned start_idx = COARSE_FACTOR * 2 * BLOCKDIM * blockIdx.x;
    unsigned tid = threadIdx.x;
    unsigned i = start_idx + tid;

    int sum = input[i];  // local var
    for (unsigned tile = 1; tile < COARSE_FACTOR * 2; tile++) {
        sum += input[i + tile * BLOCKDIM];
    }
    input_s[tid] = sum;
    __syncthreads();

    for (unsigned stride = BLOCKDIM / 2; stride > 32; stride /= 2) {
        if (tid < stride)
            input_s[tid] += input_s[tid + stride];
        __syncthreads();
    }
    
    if (tid < 32) {
        warp_reduce(input_s, tid);
    }

    if (tid == 0) {
        atomicAdd(output, input_s[0]);
    }
}

int main() {
    int *input_h = NULL;
    int *output_h = NULL;
    int *input_d = NULL;
    int *output_d = NULL;

    input_h = (int *) malloc(N * sizeof(int));
    output_h = (int *) malloc(sizeof(int));

    for (int i = 0; i < N; i++)
        input_h[i] = 1;

    hipMalloc((void **) &input_d, N * sizeof(int));
    hipMalloc((void **) &output_d, sizeof(int));

    hipMemcpy(input_d, input_h, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDim(N / BLOCKDIM / 2 / COARSE_FACTOR);
    dim3 blockDim(BLOCKDIM);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce_warp_reduce<<<gridDim, blockDim, BLOCKDIM * sizeof(int)>>>(input_d, output_d);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(output_h, output_d, sizeof(int), hipMemcpyDeviceToHost);

    int result = 0;
    for (int i = 0; i < N; i++)
        result += input_h[i];
    cout << "expected output: " << result << endl;
    cout << "output: " << *output_h << endl;
    assert(result == *output_h);
    cout << "time consumed: " << elapsedTime << "ms" << endl;

    free(input_h);
    free(output_h);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}
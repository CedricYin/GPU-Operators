#include <cassert>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#include <cfloat>

using namespace std;

// Minimizing control divergence
__global__ void reduce1(float *input, float *output) {
    const int start_idx = 2 * blockDim.x * blockIdx.x;
    const int i = start_idx + threadIdx.x;
    for (unsigned stride = blockDim.x; stride >= 1; stride /= 2) {
        if (threadIdx.x < stride)
            input[i] += input[i + stride];
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        atomicAdd(output, input[start_idx]);
    }
}

int main(int argc, char **argv) {
    constexpr int N = 8192 * 8192;
    constexpr int BLOCKDIM = 256;
    constexpr float result = N * 1.f;
    int nWarmup = 2;
    int nIters = 10;
    assert(argc == 1 || argc == 3);
    if (argc == 3) {
        nWarmup = atoi(argv[1]);
        nIters = atoi(argv[2]);
    }
    float elapsedTime;
    float *input_h = NULL;
    float *output_h = NULL;
    float *input_d = NULL;
    float *output_d = NULL;

    input_h = (float *) malloc(N * sizeof(float));
    output_h = (float *) malloc(sizeof(float));

    for (int i = 0; i < N; i++)
        input_h[i] = 1.f;

    hipMalloc((void **) &input_d, N * sizeof(float));
    hipMalloc((void **) &output_d, sizeof(float));

    hipMemcpy(input_d, input_h, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 gridDim(N / BLOCKDIM / 2);
    dim3 blockDim(BLOCKDIM);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int i = 0; i < nWarmup + nIters; i++) {
        hipEventRecord(start);
        reduce1<<<gridDim, blockDim>>>(input_d, output_d);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i == 0) {
            hipMemcpy(output_h, output_d, sizeof(float), hipMemcpyDeviceToHost);
            cout << "result: " << result << ", output: " << *output_h << '\n';
            assert(fabs(result - (*output_h)) < FLT_EPSILON);
            cout << "result is right\n";
        }
        if (i >= nWarmup) {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsedTime += ms;
        }
    }

    double bw = 4. * N / (elapsedTime / nIters / 1000) / 1e9;
    cout << "effective bandwidth: " << bw << "GB/s" << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(input_h);
    free(output_h);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}
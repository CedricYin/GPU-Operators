#include <__clang_cuda_builtin_vars.h>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>


#define N 2048

using namespace std;

// Minimizing global memory accesses
__global__ void reduce_shared(int *input, int *output) {
    unsigned i = threadIdx.x;
    extern __shared__ int input_s[];

    input_s[i] = input[i] + input[i + blockDim.x];
    for (unsigned stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (threadIdx.x < stride)
            input_s[i] += input_s[i + stride];
    }

    if (threadIdx.x == 0) {
        *output = input_s[0];
    }
}

int main() {
    int *input_h = NULL;
    int *output_h = NULL;
    int *input_d = NULL;
    int *output_d = NULL;

    input_h = (int *) malloc(N * sizeof(int));
    output_h = (int *) malloc(sizeof(int));

    for (int i = 0; i < N; i++)
        input_h[i] = 1;

    hipMalloc((void **) &input_d, N * sizeof(int));
    hipMalloc((void **) &output_d, sizeof(int));

    hipMemcpy(input_d, input_h, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDim(1);
    dim3 blockDim(N / 2);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce_shared<<<gridDim, blockDim, N / 2 * sizeof(int)>>>(input_d, output_d);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(output_h, output_d, sizeof(int), hipMemcpyDeviceToHost);

    int result = 0;
    for (int i = 0; i < N; i++)
        result += input_h[i];
    cout << "expected output: " << result << endl;
    cout << "output: " << *output_h << endl;
    assert(result == *output_h);
    cout << "time consumed: " << elapsedTime << "ms" << endl;

    free(input_h);
    free(output_h);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}
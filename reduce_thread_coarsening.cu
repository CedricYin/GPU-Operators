#include <cassert>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>


#define N 256 * 1024 * 1024
#define BLOCKDIM 1024
#define COARSE_FACTOR 2

using namespace std;

// Thread coarsening for reduced overhead
__global__ void reduce_thread_coarsening(int *input, int *output) {
    extern __shared__ int input_s[];
    unsigned start_idx = COARSE_FACTOR * 2 * BLOCKDIM * blockIdx.x;
    unsigned tid = threadIdx.x;
    unsigned i = start_idx + tid;

    input_s[tid] = input[i];
    for (unsigned tile = 1; tile < COARSE_FACTOR * 2; tile++) {
        input_s[tid] += input[i + tile * BLOCKDIM];
    }

    for (unsigned stride = BLOCKDIM / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride)
            input_s[tid] += input_s[tid + stride];
    }

    if (tid == 0) {
        atomicAdd(output, input_s[0]);
    }
}

int main() {
    int *input_h = NULL;
    int *output_h = NULL;
    int *input_d = NULL;
    int *output_d = NULL;

    input_h = (int *) malloc(N * sizeof(int));
    output_h = (int *) malloc(sizeof(int));

    for (int i = 0; i < N; i++)
        input_h[i] = 1;

    hipMalloc((void **) &input_d, N * sizeof(int));
    hipMalloc((void **) &output_d, sizeof(int));

    hipMemcpy(input_d, input_h, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDim(N / BLOCKDIM / 2 / COARSE_FACTOR);
    dim3 blockDim(BLOCKDIM);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce_thread_coarsening<<<gridDim, blockDim, BLOCKDIM * sizeof(int)>>>(input_d, output_d);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(output_h, output_d, sizeof(int), hipMemcpyDeviceToHost);

    int result = 0;
    for (int i = 0; i < N; i++)
        result += input_h[i];
    cout << "expected output: " << result << endl;
    cout << "output: " << *output_h << endl;
    assert(result == *output_h);
    cout << "time consumed: " << elapsedTime << "ms" << endl;

    free(input_h);
    free(output_h);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

// 0.816320 ms
__global__ void copy_if0(int * dst, const int *src, int *len, const uint N) {
  const uint i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < N && src[i] > 0)
    dst[atomicAdd(len, 1)] = src[i];
}

bool CheckResult(int *out, int groudtruth, int n){
    if (*out != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    int N = 25600000;
    const int blockSize = 256;
    int GridSize = (int) ceil(1.f*N/blockSize);

    int *src_h = (int *)malloc(N * sizeof(int));
    int *dst_h = (int *)malloc(N * sizeof(int));
    int *nres_h = (int *)malloc(1 * sizeof(int));
    int *dst, *nres;
    int *src;
    hipMalloc((void **)&src, N * sizeof(int));
    hipMalloc((void **)&dst, N * sizeof(int));
    hipMalloc((void **)&nres, 1 * sizeof(int));

    for(int i = 0; i < N; i++){
        src_h[i] = 1;
    }

    int groudtruth = 0;
    for(int j = 0; j < N; j++){
        if (src_h[j] > 0) {
            groudtruth += 1;
        }
    }


    hipMemcpy(src, src_h, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    copy_if0<<<Grid, Block>>>(dst, src, nres, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(nres_h, nres, 1 * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(nres_h, groudtruth, N);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        printf("%d ",*nres_h);
        printf("\n");
    }
    printf("filter_k latency = %f ms\n", milliseconds);    

    hipFree(src);
    hipFree(dst);
    hipFree(nres);
    free(src_h);
    free(dst_h);
    free(nres_h);
}
#include <cstddef>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>
#include <cstdlib>

using namespace std;

#define M 1024
#define N 1024
#define K 1024
#define BLOCKDIM 32
#define TILEDIM 32

// A: M * K; B: K * N
__global__ void sgemm_v1(float *a, float *b, float *c) {
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    int grow = by * blockDim.y + ty;
    int gcol = bx * blockDim.x + tx;
    __shared__ float tileA[TILEDIM][TILEDIM];
    __shared__ float tileB[TILEDIM][TILEDIM];

    int phase = ceil(1.f * K / TILEDIM);
    float pval = 0.f;
    for (int i = 0; i < phase; i++) {
        // global -> shared: load tile
        if (grow < M && i * TILEDIM + tx < K)
            tileA[ty][tx] = a[grow * K + i * TILEDIM + tx];
        else
            tileA[ty][tx] = 0.f;
        if (i * TILEDIM + ty < K && gcol < N)
            tileB[ty][tx] = b[(i * TILEDIM + ty) * N + gcol];
        else
            tileB[ty][tx] = 0.f;
        __syncthreads();

        // partial dot product
        for (int k = 0; k < TILEDIM; k++) {
            pval += tileA[ty][k] * tileB[k][tx];
        }
        __syncthreads();
    }

    if (grow < M && gcol < N) {
        c[grow * N + gcol] = pval;
    }
}

float* init(float *a, float *b) {
    for (int i = 0; i < M * N; i++) {
        a[i] = 1.f;
        b[i] = 1.f;
    }
    float *c = (float *) calloc(M * N, sizeof(float));
    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++)
            for (int k = 0; k < K; k++)
                c[i * N + j] += a[i * K + k] * b[k * N + j];
    
    return c;
}

bool check_ans(float *truth, float *c) {
    for (int i = 0; i < M * N; i++)
        if (fabs(c[i] - truth[i]) > 0.5) {
            printf("truth: %f, output: %f\n", truth[i], c[i]);
            return false;
        }
    return true;
}

int main() {
    size_t size = sizeof(float) * M * N;
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    float *c_truth = init(a_h, b_h);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed = 0.f;
    const int nWarmup = 2;
    const int nIters = 3;

    dim3 gridDim(ceil(1.f * M / BLOCKDIM), ceil(1.f * N / BLOCKDIM), 1);
    dim3 blockDim(BLOCKDIM, BLOCKDIM, 1);  // block 的y维缩小COARSENING_FACTOR倍

    for (int i = 0; i < nIters + nWarmup; i++) {
        hipEventRecord(start);
        sgemm_v1<<<gridDim, blockDim>>>(a_d, b_d, c_d);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i < nWarmup) {
            if (i == 0) {
                hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
                if (!check_ans(c_truth, c_h)) {
                    cerr << "result is wrong!" << endl;
                    return -1;
                }
                cout << "result is right" << endl;
            }
        } else {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsed += ms;
        }
    }
    const int64_t flop = int64_t(M) * int64_t(N) * int64_t(K) * 2;
    double gflops = flop / ((elapsed / nIters) / 1000) / 1e9;
    cout << "kernel: " << gflops << "GFLOPS (" << flop << " flop, " << (elapsed / nIters) / 1000 << "s)\n";

    return 0;
}
#include <cstddef>
#include <cassert>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <sys/cdefs.h>
#include <hipblas.h>

using namespace std;

template<int TM, int TN, int TK>
__global__ void sgemm_v1(const float *__restrict__ A, const float *__restrict__ B, float *C, 
                        int M, int N, int K,
                        float alpha, float beta) {
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    __shared__ float As[TM][TK];
    __shared__ float Bs[TK][TN];

    // 移动到当前C block，以及A和B的起始位置
    A = &A[by * TM * K];
    B = &B[bx * TN];
    C = &C[by * TM * N + bx * TN];

    float pval = 0.f;
    for (int k = 0; k < K; k += TK) {
        // global to shared
        if (ty < TM && tx < TK)  // 必须加判断，否则会段错误。因为此时block里的线程数量比As里的元素数量多，下面Bs同理
            As[ty][tx] = A[ty * K + tx];
        if (ty < TK && tx < TN)
            Bs[ty][tx] = B[ty * N + tx];
        __syncthreads();

        // 移动到下一个迭代的位置
        A += TK;
        B += TK * N;

        // partial dot product
        for (int i = 0; i < TK; i++) {
            pval += As[ty][i] * Bs[i][tx];
        }
        // FMA计算需要读取缓存数据，在新一轮写入缓存前进行同步，确保所有线程计算完成
        __syncthreads();
    }
    C[ty * N + tx] = alpha * pval + beta * C[ty * N + tx];
}

void init(float *a, float *b, int M, int N, int K) {
    for (int i = 0; i < M * N; i++) {
        a[i] = 1.f;
        b[i] = 1.f;
    }
}

bool check_ans(float *truth, float *c, int M, int N) {
    for (int i = 0; i < M * N; i++)
        if (fabs(c[i] - truth[i]) > 0.5) {
            printf("truth: %f, output: %f\n", truth[i], c[i]);
            return false;
        }
    return true;
}

int main(int argc, char **argv) {
    // ncu: ./sgemm 0 1
    // compute-santi: ./sgemm 0 1 256 256 128
    // v100: ./sgemm
    int M = 8192;
    int N = 8192;
    int K = 4096;
    constexpr int TM = 32;
    constexpr int TN = 32;
    constexpr int TK = 32;
    int nWarmup = 2;
    int nIters = 50;
    assert(argc == 1 || argc == 3 || argc == 6);
    if (argc >= 3) {
        nWarmup = atoi(argv[1]);
        nIters = atoi(argv[2]);
    }
    if (argc >= 6) {
        M = atoi(argv[3]);
        N = atoi(argv[4]);
        K = atoi(argv[5]);
    }
    const float alpha = 1.f;
    const float beta = 0.f;
    float elapsed_my = 0.f;
    float elapsed_cublas = 0.f;
    size_t size = sizeof(float) * M * N;
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    float *c_truth = (float *) malloc(size);
    init(a_h, b_h, M, N, K);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.f * N / TN), ceil(1.f * M / TM), 1);
    dim3 blockDim(TN, TM, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // my sgemm
    for (int i = 0; i < nIters + nWarmup; i++) {
        hipEventRecord(start);
        sgemm_v1<TM, TN, TK><<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K, alpha, beta);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i >= nWarmup) {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsed_my += ms;
        }
    }
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    #ifndef PROFILE
    // cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    for (int i = 0; i < nIters; i++) {
        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b_d, N, a_d, K, &beta, c_d, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        elapsed_cublas += ms;
    }
    hipblasDestroy(handle);
    hipMemcpy(c_truth, c_d, size, hipMemcpyDeviceToHost);

    // check
    if (!check_ans(c_truth, c_h, M, N)) {
        cerr << "result is wrong!" << endl;
        return -1;
    }
    cout << "result is right" << endl;

    // output
    const int64_t flop = int64_t(M) * int64_t(N) * int64_t(K) * 2;
    double gflops_my = flop / ((elapsed_my / nIters) / 1000) / 1e9;
    double gflops_cublas = flop / ((elapsed_cublas / nIters) / 1000) / 1e9;
    cout << "mysgemm: " << gflops_my << "GFLOPS (" << flop << " flop, " << (elapsed_my / nIters) / 1000 << "s)\n";
    cout << "cublas: " << gflops_cublas << "GFLOPS (" << flop << " flop, " << (elapsed_cublas / nIters) / 1000 << "s)\n";
    cout << "% of cublas: " << gflops_my / gflops_cublas * 100 << "%" << endl;
    #endif

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a_h);
    free(b_h);
    free(c_h);
    free(c_truth);

    return 0;
}
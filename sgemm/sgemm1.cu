#include <cstddef>
#include <cassert>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <sys/cdefs.h>
#include <hipblas.h>

using namespace std;

template<int TILEDIM>
__global__ void sgemm_v1(const float *__restrict__ a, const float *__restrict__ b, float *c, 
                        int M, int N, int K,
                        float alpha, float beta) {
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    int grow = by * blockDim.y + ty;
    int gcol = bx * blockDim.x + tx;
    __shared__ float tileA[TILEDIM][TILEDIM];
    __shared__ float tileB[TILEDIM][TILEDIM];

    int phase = ceil(1.f * K / TILEDIM);
    float pval = 0.f;
    for (int i = 0; i < phase; i++) {
        // global -> shared: load tile
        if (grow < M && i * TILEDIM + tx < K)
            tileA[ty][tx] = a[grow * K + i * TILEDIM + tx];
        else
            tileA[ty][tx] = 0.f;
        if (i * TILEDIM + ty < K && gcol < N)
            tileB[ty][tx] = b[(i * TILEDIM + ty) * N + gcol];
        else
            tileB[ty][tx] = 0.f;
        __syncthreads();

        // partial dot product
        for (int k = 0; k < TILEDIM; k++) {
            pval += tileA[ty][k] * tileB[k][tx];
        }
        __syncthreads();
    }

    if (grow < M && gcol < N) {
        c[grow * N + gcol] = alpha * pval + beta * c[grow * N + gcol];
    }
}

void init(float *a, float *b, int M, int N, int K) {
    for (int i = 0; i < M * N; i++) {
        a[i] = 1.f;
        b[i] = 1.f;
    }
}

bool check_ans(float *truth, float *c, int M, int N) {
    for (int i = 0; i < M * N; i++)
        if (fabs(c[i] - truth[i]) > 0.5) {
            printf("truth: %f, output: %f\n", truth[i], c[i]);
            return false;
        }
    return true;
}

int main(int argc, char **argv) {
    // ncu: ./sgemm 0 1
    // compute-santi: ./sgemm 0 1 256 256 128
    // v100: ./sgemm
    int M = 8192;
    int N = 8192;
    int K = 4096;
    int nWarmup = 2;
    int nIters = 50;
    assert(argc == 1 || argc == 3 || argc == 6);
    if (argc >= 3) {
        nWarmup = atoi(argv[1]);
        nIters = atoi(argv[2]);
    }
    if (argc >= 6) {
        M = atoi(argv[3]);
        N = atoi(argv[4]);
        K = atoi(argv[5]);
    }
    const float alpha = 1.f;
    const float beta = 0.f;
    const int BLOCKDIM = 32;
    const int TILEDIM = 32;
    float elapsed_my = 0.f;
    float elapsed_cublas = 0.f;
    size_t size = sizeof(float) * M * N;
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    float *c_truth = (float *) malloc(size);
    init(a_h, b_h, M, N, K);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.f * M / BLOCKDIM), ceil(1.f * N / BLOCKDIM), 1);
    dim3 blockDim(BLOCKDIM, BLOCKDIM, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // my sgemm
    for (int i = 0; i < nIters + nWarmup; i++) {
        hipEventRecord(start);
        sgemm_v1<TILEDIM><<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K, alpha, beta);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i >= nWarmup) {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsed_my += ms;
        }
    }
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    #ifndef PROFILE
    // cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    for (int i = 0; i < nIters; i++) {
        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b_d, N, a_d, K, &beta, c_d, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        elapsed_cublas += ms;
    }
    hipblasDestroy(handle);
    hipMemcpy(c_truth, c_d, size, hipMemcpyDeviceToHost);

    // check
    if (!check_ans(c_truth, c_h, M, N)) {
        cerr << "result is wrong!" << endl;
        return -1;
    }
    cout << "result is right" << endl;

    // output
    const int64_t flop = int64_t(M) * int64_t(N) * int64_t(K) * 2;
    double gflops_my = flop / ((elapsed_my / nIters) / 1000) / 1e9;
    double gflops_cublas = flop / ((elapsed_cublas / nIters) / 1000) / 1e9;
    cout << "mysgemm: " << gflops_my << "GFLOPS (" << flop << " flop, " << (elapsed_my / nIters) / 1000 << "s)\n";
    cout << "cublas: " << gflops_cublas << "GFLOPS (" << flop << " flop, " << (elapsed_cublas / nIters) / 1000 << "s)\n";
    cout << "% of cublas: " << gflops_my / gflops_cublas * 100 << "%" << endl;
    #endif

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a_h);
    free(b_h);
    free(c_h);
    free(c_truth);

    return 0;
}
#include <cstddef>
#include <cassert>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <sys/cdefs.h>
#include <hipblas.h>

#define OFFSET(row, col, stride) ((row) * (stride) + (col))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

using namespace std;

template<int TM, int TN, int TK, int RM, int RN>
__global__ void sgemm_v3(float *__restrict__ A, float *__restrict__ B, float *C, 
                        int M, int N, int K,
                        float alpha, float beta) {
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    constexpr int thread_nums = (TM / RM) * (TN / RN);
    // (y_st, x_st) 是每个线程所负责的RM*RN块的左上角坐标
    const int x_st = (threadIdx.x % (TN / RN)) * RN;
    const int y_st = (threadIdx.x / (TM / RM)) * RM;
    
    __shared__ float As[TK][TM];  // 为了便于float4读取，这里As进行转置
    __shared__ float Bs[TK][TN];

    // 移动到当前要处理的C block，以及A和B的起始位置
    A = &A[by * TM * K];
    B = &B[bx * TN];
    C = &C[by * TM * N + bx * TN];

    // 重新将一维的线程组织成二维的:
    // 组织成有TK列的线程，用来搬运数据到As
    const int a_tile_y = threadIdx.x / (TK / 4);
    const int a_tile_x = threadIdx.x % (TK / 4) * 4;
    constexpr int a_tile_stride = thread_nums / (TK / 4); // 每轮跨越的行数
    // 组织成有TN列的线程，用来搬运数据到Bs
    const int b_tile_y = threadIdx.x / (TN / 4);
    const int b_tile_x = threadIdx.x % (TN / 4) * 4;
    constexpr int b_tile_stride = thread_nums / (TN / 4);

    float pval[RM][RN] = {0.f};  // 每个线程负责RM*RN个位置
    // float a_tmp[4 * (TM * TK) / thread_nums / 4] = {0.f};
    
    for (int phase = 0; phase < K; phase += TK) {
        // global to shared
        for (int i = 0; i < TM; i += a_tile_stride) {
            // int a_tmp_idx = i / a_tile_stride * 4;
            // FETCH_FLOAT4(a_tmp[a_tmp_idx]) = FETCH_FLOAT4(A[OFFSET(a_tile_y + i, a_tile_x, K)]);
            // As[a_tile_x][a_tile_y + i] = a_tmp[a_tmp_idx];
            // As[a_tile_x + 1][a_tile_y + i] = a_tmp[a_tmp_idx + 1];
            // As[a_tile_x + 2][a_tile_y + i] = a_tmp[a_tmp_idx + 2];
            // As[a_tile_x + 3][a_tile_y + i] = a_tmp[a_tmp_idx + 3];
            float a_tmp[4] = {0.f};
            FETCH_FLOAT4(a_tmp[0]) = FETCH_FLOAT4(A[OFFSET(a_tile_y + i, a_tile_x, K)]);
            As[a_tile_x][a_tile_y + i] = a_tmp[0];
            As[a_tile_x + 1][a_tile_y + i] = a_tmp[1];
            As[a_tile_x + 2][a_tile_y + i] = a_tmp[2];
            As[a_tile_x + 3][a_tile_y + i] = a_tmp[3];
        }
        for (int i = 0; i < TK; i += b_tile_stride) {
            FETCH_FLOAT4(Bs[b_tile_y + i][b_tile_x]) = FETCH_FLOAT4(B[OFFSET(b_tile_y + i, b_tile_x, N)]);
        }
        __syncthreads();

        // 移动到下一个迭代的位置
        A += TK;
        B += TK * N;

        // partial dot product
        for (int k = 0; k < TK; k++) {
            // shared to register
            float Areg[RM], Breg[RN];
            for (int m = 0; m < RM; m += 4) FETCH_FLOAT4(Areg[m]) = FETCH_FLOAT4(As[k][y_st + m]);
            for (int n = 0; n < RN; n += 4) FETCH_FLOAT4(Breg[n]) = FETCH_FLOAT4(Bs[k][x_st + n]);

            for (int m = 0; m < RM; m++)
                for (int n = 0; n < RN; n++)
                    pval[m][n] += Areg[m] * Breg[n];
        }
        __syncthreads();
    }

    float c_tmp[4] = {0.f};
    for (int m = 0; m < RM; m++) {
        for (int n = 0; n < RN; n += 4) {
            FETCH_FLOAT4(c_tmp[0]) = FETCH_FLOAT4(C[OFFSET(y_st + m, x_st + n, N)]);
            c_tmp[0] = alpha * pval[m][n] + beta * c_tmp[0];
            c_tmp[1] = alpha * pval[m][n + 1] + beta * c_tmp[1];
            c_tmp[2] = alpha * pval[m][n + 2] + beta * c_tmp[2];
            c_tmp[3] = alpha * pval[m][n + 3] + beta * c_tmp[3];
            FETCH_FLOAT4(C[OFFSET(y_st + m, x_st + n, N)]) = FETCH_FLOAT4(c_tmp[0]);
        }
    }
}

void init(float *a, float *b, int M, int N, int K) {
    for (int i = 0; i < M * N; i++) {
        a[i] = 1.f;
        b[i] = 1.f;
    }
}

bool check_ans(float *truth, float *c, int M, int N) {
    for (int i = 0; i < M * N; i++)
        if (fabs(c[i] - truth[i]) > 0.5) {
            printf("truth: %f, output: %f\n", truth[i], c[i]);
            return false;
        }
    return true;
}

int main(int argc, char **argv) {
    // ncu: ./sgemm 0 1
    // compute-santi: ./sgemm 0 1 256 256 128
    // v100: ./sgemm
    int M = 8192;
    int N = 8192;
    int K = 4096;
    constexpr int TM = 128;
    constexpr int TN = 128;
    constexpr int TK = 8;
    constexpr int RM = 8;
    constexpr int RN = 8;
    int nWarmup = 2;
    int nIters = 50;
    assert(argc == 1 || argc == 3 || argc == 6);
    if (argc >= 3) {
        nWarmup = atoi(argv[1]);
        nIters = atoi(argv[2]);
    }
    if (argc >= 6) {
        M = atoi(argv[3]);
        N = atoi(argv[4]);
        K = atoi(argv[5]);
    }
    const float alpha = 1.f;
    const float beta = 0.f;
    float elapsed_my = 0.f;
    float elapsed_cublas = 0.f;
    size_t size = sizeof(float) * M * N;
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    float *c_truth = (float *) malloc(size);
    init(a_h, b_h, M, N, K);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.f * N / TN), ceil(1.f * M / TM), 1);
    dim3 blockDim((TN / RN) * (TM / RM), 1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // my sgemm
    for (int i = 0; i < nIters + nWarmup; i++) {
        hipEventRecord(start);
        sgemm_v3<TM, TN, TK, RM, RN><<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K, alpha, beta);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i >= nWarmup) {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsed_my += ms;
        }
    }
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    #ifndef PROFILE
    // cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    for (int i = 0; i < nIters; i++) {
        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b_d, N, a_d, K, &beta, c_d, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        elapsed_cublas += ms;
    }
    hipblasDestroy(handle);
    hipMemcpy(c_truth, c_d, size, hipMemcpyDeviceToHost);

    // check
    if (!check_ans(c_truth, c_h, M, N)) {
        cerr << "result is wrong!" << endl;
        return -1;
    }
    cout << "result is right" << endl;

    // output
    const int64_t flop = int64_t(M) * int64_t(N) * int64_t(K) * 2;
    double gflops_my = flop / ((elapsed_my / nIters) / 1000) / 1e9;
    double gflops_cublas = flop / ((elapsed_cublas / nIters) / 1000) / 1e9;
    cout << "mysgemm: " << gflops_my << "GFLOPS (" << flop << " flop, " << (elapsed_my / nIters) / 1000 << "s)\n";
    cout << "cublas: " << gflops_cublas << "GFLOPS (" << flop << " flop, " << (elapsed_cublas / nIters) / 1000 << "s)\n";
    cout << "% of cublas: " << gflops_my / gflops_cublas * 100 << "%" << endl;
    #endif

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a_h);
    free(b_h);
    free(c_h);
    free(c_truth);

    return 0;
}
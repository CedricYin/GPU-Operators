#include <cstddef>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <sys/cdefs.h>
#include <hipblas.h>

using namespace std;

// A: M * K; B: K * N
__global__ void sgemm_v0(const float *__restrict__ a, const float *__restrict__ b, float *c, 
                        int M, int N, int K,
                        float alpha, float beta) {
    int grow = blockDim.y * blockIdx.y + threadIdx.y;
    int gcol = blockDim.x * blockIdx.x + threadIdx.x;

    if (grow < M && gcol < N) {
        float val = 0.f;
        for (int k = 0; k < K; k++)
            val += a[grow * K + k] * b[k * N + gcol];

        c[grow * N + gcol] = val;
    }
}

void init(float *a, float *b, int M, int N, int K) {
    for (int i = 0; i < M * N; i++) {
        a[i] = 1.f;
        b[i] = 1.f;
    }
}

bool check_ans(float *truth, float *c, int M, int N) {
    for (int i = 0; i < M * N; i++)
        if (fabs(c[i] - truth[i]) > 0.5) {
            printf("truth: %f, output: %f\n", truth[i], c[i]);
            return false;
        }
    return true;
}

int main() {
    const int M = 8192;
    const int N = 8192;
    const int K = 4096;
    const float alpha = 1.f;
    const float beta = 0.f;
    const int BLOCKDIM = 32;
    float elapsed_my = 0.f;
    float elapsed_cublas = 0.f;
    const int nWarmup = 2;
    const int nIters = 50;
    size_t size = sizeof(float) * M * N;
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    float *c_truth = (float *) malloc(size);
    init(a_h, b_h, M, N, K);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.f * M / BLOCKDIM), ceil(1.f * N / BLOCKDIM), 1);
    dim3 blockDim(BLOCKDIM, BLOCKDIM, 1);  // block 的y维缩小COARSENING_FACTOR倍

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // my sgemm
    for (int i = 0; i < nIters + nWarmup; i++) {
        hipEventRecord(start);
        sgemm_v0<<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K, alpha, beta);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i >= nWarmup) {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsed_my += ms;
        }
    }
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    // cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    for (int i = 0; i < 50; i++) {
        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b_d, N, a_d, K, &beta, c_d, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        elapsed_cublas += ms;
    }
    hipMemcpy(c_truth, c_d, size, hipMemcpyDeviceToHost);

    // check
    if (!check_ans(c_truth, c_h, M, N)) {
        cerr << "result is wrong!" << endl;
        return -1;
    }
    cout << "result is right" << endl;

    // output
    const int64_t flop = int64_t(M) * int64_t(N) * int64_t(K) * 2;
    double gflops_my = flop / ((elapsed_my / nIters) / 1000) / 1e9;
    double gflops_cublas = flop / ((elapsed_cublas / 50) / 1000) / 1e9;
    cout << "mysgemm: " << gflops_my << "GFLOPS (" << flop << " flop, " << (elapsed_my / nIters) / 1000 << "s)\n";
    cout << "cublas: " << gflops_cublas << "GFLOPS (" << flop << " flop, " << (elapsed_cublas / 50) / 1000 << "s)\n";
    cout << "% of cublas: " << gflops_my / gflops_cublas * 100 << "%" << endl;

    return 0;
}
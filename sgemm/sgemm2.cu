#include <cstddef>
#include <cassert>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <sys/cdefs.h>
#include <hipblas.h>

using namespace std;

template<int TILEDIM, int COARSENING_FACTOR, int PATCH>
__global__ void sgemm_v2(const float *__restrict__ a, const float *__restrict__ b, float *c, 
                        int M, int N, int K,
                        float alpha, float beta) {
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    int grow = by * blockDim.y * COARSENING_FACTOR + ty;
    int gcol = bx * blockDim.x + tx;
    __shared__ float tileA[TILEDIM][TILEDIM];
    __shared__ float tileB[TILEDIM][TILEDIM];

    int phase = ceil(1.f * K / TILEDIM);
    float pval[COARSENING_FACTOR] = {0.f};
    for (int i = 0; i < phase; i++) {
        // global -> shared: load tile
        #pragma unroll
        for (int j = 0; j < COARSENING_FACTOR; j++) {
            if (grow + j * PATCH < M && i * TILEDIM + tx < K)
                tileA[ty + j * PATCH][tx] = a[(grow + j * PATCH) * K + i * TILEDIM + tx];
            else
                tileA[ty + j * PATCH][tx] = 0.f;
        }
        #pragma unroll
        for (int j = 0; j < COARSENING_FACTOR; j++) {
            if (i * TILEDIM + ty + j * PATCH < K && gcol < N)
                tileB[ty + j * PATCH][tx] = b[(i * TILEDIM + ty + j * PATCH) * N + gcol];
            else 
                tileB[ty + j * PATCH][tx] = 0.f;
        }
        __syncthreads();

        // partial dot product
        for (int k = 0; k < TILEDIM; k++) {
            float reg_b = tileB[k][tx];
            #pragma unroll
            for (int p = 0; p < COARSENING_FACTOR; p++) {
                pval[p] += tileA[ty + p * PATCH][k] * reg_b;  // register value can be reused multiple times
            }
        }
        __syncthreads();
    }

    #pragma unroll
    for (int i = 0; i < COARSENING_FACTOR; i++) {
        if (grow + i * PATCH < M && gcol < N)
            c[(grow + i * PATCH) * N + gcol] = alpha * pval[i] + beta * c[(grow + i * PATCH) * N + gcol];
    }
}

void init(float *a, float *b, int M, int N, int K) {
    for (int i = 0; i < M * N; i++) {
        a[i] = 1.f;
        b[i] = 1.f;
    }
}

bool check_ans(float *truth, float *c, int M, int N) {
    for (int i = 0; i < M * N; i++)
        if (fabs(c[i] - truth[i]) > 0.5) {
            printf("truth: %f, output: %f\n", truth[i], c[i]);
            return false;
        }
    return true;
}

int main(int argc, char **argv) {
    // ncu: ./sgemm 0 1
    // compute-santi: ./sgemm 0 1 256 256 128
    // v100: ./sgemm
    int M = 8192;
    int N = 8192;
    int K = 4096;
    int nWarmup = 2;
    int nIters = 50;
    assert(argc == 1 || argc == 3 || argc == 6);
    if (argc >= 3) {
        nWarmup = atoi(argv[1]);
        nIters = atoi(argv[2]);
    }
    if (argc >= 6) {
        M = atoi(argv[3]);
        N = atoi(argv[4]);
        K = atoi(argv[5]);
    }
    const float alpha = 1.f;
    const float beta = 0.f;
    const int BLOCKDIM = 32;
    const int TILEDIM = 32;
    const int COARSENING_FACTOR = 2;
    float elapsed_my = 0.f;
    float elapsed_cublas = 0.f;
    size_t size = sizeof(float) * M * N;
    float *a_h = (float *) malloc(size);
    float *b_h = (float *) malloc(size);
    float *c_h = (float *) malloc(size);
    float *c_truth = (float *) malloc(size);
    init(a_h, b_h, M, N, K);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.f * M / BLOCKDIM), ceil(1.f * N / BLOCKDIM), 1);
    dim3 blockDim(BLOCKDIM, BLOCKDIM / COARSENING_FACTOR, 1);  // block 的y维缩小COARSENING_FACTOR倍

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // my sgemm
    for (int i = 0; i < nIters + nWarmup; i++) {
        hipEventRecord(start);
        sgemm_v2<TILEDIM, COARSENING_FACTOR, BLOCKDIM / COARSENING_FACTOR><<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K, alpha, beta);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        if (i >= nWarmup) {
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << i - nWarmup << ": " << ms << " ms\n";
            elapsed_my += ms;
        }
    }
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    #ifndef PROFILE
    // cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    for (int i = 0; i < nIters; i++) {
        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b_d, N, a_d, K, &beta, c_d, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        elapsed_cublas += ms;
    }
    hipblasDestroy(handle);
    hipMemcpy(c_truth, c_d, size, hipMemcpyDeviceToHost);

    // check
    if (!check_ans(c_truth, c_h, M, N)) {
        cerr << "result is wrong!" << endl;
        return -1;
    }
    cout << "result is right" << endl;

    // output
    const int64_t flop = int64_t(M) * int64_t(N) * int64_t(K) * 2;
    double gflops_my = flop / ((elapsed_my / nIters) / 1000) / 1e9;
    double gflops_cublas = flop / ((elapsed_cublas / nIters) / 1000) / 1e9;
    cout << "mysgemm: " << gflops_my << "GFLOPS (" << flop << " flop, " << (elapsed_my / nIters) / 1000 << "s)\n";
    cout << "cublas: " << gflops_cublas << "GFLOPS (" << flop << " flop, " << (elapsed_cublas / nIters) / 1000 << "s)\n";
    cout << "% of cublas: " << gflops_my / gflops_cublas * 100 << "%" << endl;
    #endif

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a_h);
    free(b_h);
    free(c_h);
    free(c_truth);

    return 0;
}